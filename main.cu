
#include <hip/hip_runtime.h>
#include <iostream>

#define N 32 // blockDim.x = blockDim.y = N 
#define X (threadIdx.x) // to simplify expressions 
#define Y (threadIdx.y) // to simplify expressions 
#define RANGE_OF_SEARCH_SHRINK 0.5f

__global__ void kernel(float * best_corr, float * best_offset, float * best_scale, const float * const experiment, const float * const theory_lib, int data_size) {
    // resolution of both theory and experiment is 1 nm per point. So, the total extension = data_size
    __shared__ float cache_correlation[N][N];
    __shared__ int cache_offset_idx[N][N];
    __shared__ int cache_scale_idx[N][N];


    float min_scale = 0.9, max_scale = 1.1;
    float min_offset = -500, max_offset =500;
    
    // The surface has local minima so I cannot use general minimization method, for example downhill simplex.
    // I have to use a brute force search, after each loop I shrink the search area a little bit.
    while(true) {
        // reset cache for this thread, the values will be changed during reduction to same the max_correlations' threadIdx.x and threadIdx.y
        cache_offset_idx[X][Y] = X;
        cache_scale_idx [X][Y] = Y;
        // set offset and scale of this thread
        float offset = (float)X/(N - 1) * (max_offset - min_offset) + min_offset;
        float scale  = (float)Y/(N - 1) * (max_scale  - min_scale)  + min_scale;

        // Calculate the correlation:
        float act_x = experiment[0] + theory_lib[blockIdx.x * data_size + 0] + offset + scale; //actual extension
        cache_correlation[X][Y] = (float)1000 - (X - N/2 + blockIdx.x) * (X - N/2 + blockIdx.x) - (Y - N/2, 2) * (Y - N/2, 2); // placeholder
        // if(X == 15 && Y == 15) {
        //     printf("~In kernel print, block %d, corr[%d][%d] = %f.\n", blockIdx.x, 15, 15, cache_correlation[15][15]);
        // }

        __syncthreads(); // wait for all correlation is calculated

        
        // My note for O(log n) redution of 2d array: because this is a 2D array, I have to compare 4 values each time in 4 smaller submatrices. 
        // Find max of each row has a time complexity of O(nlogn), therefore not right. 
        // I also need 2 arrays to track the argx and argy of max correlation. 
        float max1, xmax1, ymax1, max2, xmax2, ymax2;
        int i = blockDim.x/2;
        while (i != 0) {
            if (X < i && Y < i) {
                max1 =  cache_correlation[X][Y] > cache_correlation[X][Y + i] ? cache_correlation[X][Y] : cache_correlation[X][Y + i];
                xmax1 = cache_correlation[X][Y] > cache_correlation[X][Y + i] ? cache_offset_idx [X][Y] : cache_offset_idx [X][Y + i];
                ymax1 = cache_correlation[X][Y] > cache_correlation[X][Y + i] ? cache_scale_idx  [X][Y] : cache_scale_idx  [X][Y + i];

                
                max2 =  cache_correlation[X + i][Y] > cache_correlation[X + i][Y + i] ? cache_correlation[X + i][Y] : cache_correlation[X + i][Y + i];
                xmax2 = cache_correlation[X + i][Y] > cache_correlation[X + i][Y + i] ? cache_offset_idx [X + i][Y] : cache_offset_idx [X + i][Y + i];
                ymax2 = cache_correlation[X + i][Y] > cache_correlation[X + i][Y + i] ? cache_scale_idx  [X + i][Y] : cache_scale_idx  [X + i][Y + i];

                cache_correlation[X][Y] = max1 > max2 ? max1 : max2;
                cache_offset_idx [X][Y] = max1 > max2 ? xmax1: xmax2;
                cache_scale_idx  [X][Y] = max1 > max2 ? ymax1: ymax2;
            }
            __syncthreads();
            i >>= 1;
        }
        
        break;

        // adjust the offset and scale search range by half and repeat find the global max correlation
        max_offset = cache_offset_idx[0][0] + RANGE_OF_SEARCH_SHRINK * 0.5 * (max_offset - min_offset);
        min_offset = cache_offset_idx[0][0] - RANGE_OF_SEARCH_SHRINK * 0.5 * (max_offset - min_offset);
        max_scale  = cache_scale_idx [0][0] + RANGE_OF_SEARCH_SHRINK * 0.5 * (max_scale -  min_scale);
        min_scale  = cache_scale_idx [0][0] - RANGE_OF_SEARCH_SHRINK * 0.5 * (max_scale -  min_scale);
    }


    best_corr  [blockIdx.x] = cache_correlation[0][0];
    best_offset[blockIdx.x] = (float) cache_offset_idx[0][0]/(N - 1) * (max_offset - min_offset) + min_offset;
    best_scale [blockIdx.x] = (float) cache_scale_idx [0][0]/(N - 1) * (max_scale -  min_scale)  + min_scale;

        
    if(X == 0 && Y == 0) {
        printf("In kernel print, block %d, corr[%d][%d] = %f.\n", blockIdx.x, cache_offset_idx[0][0], cache_scale_idx[0][0], cache_correlation[0][0]);
    }
}

int main() {
    
    dim3 blockDim(16, 16);
    // placeholder;
    int Nblock = 10;//how many genes in the library
    int trace_length = 100;
    float * experiment = new float[trace_length]; 
    float * theory_lib = new float [Nblock * trace_length];
    // pad the empty data with -1.0f
    for (int i = 0; i < trace_length; ++i) {
        experiment[i] = -1.0f;
        for(int j = 0; j < Nblock; ++j) {
            theory_lib[i * Nblock + j] = -1.0f;
        }
    }

    float * d_experiment, * d_theory_lib;
    hipMalloc((void**)&d_experiment, sizeof(float) * Nblock);
    hipMalloc((void**)&d_theory_lib, sizeof(float) * Nblock * trace_length);
    hipMemcpy(d_experiment, experiment, sizeof(float) * trace_length, hipMemcpyHostToDevice);
    hipMemcpy(d_theory_lib, theory_lib, sizeof(float) * Nblock * trace_length, hipMemcpyHostToDevice);
    // end placeholder

    float * d_best_corr, * d_best_offset, * d_best_scale;
    hipMalloc((void**)&d_best_corr,   sizeof(float) * Nblock);
    hipMalloc((void**)&d_best_offset, sizeof(float) * Nblock);
    hipMalloc((void**)&d_best_scale,  sizeof(float) * Nblock);

    

    // float * best_corr, float * best_scale, float * best_offset, float * experiment, float ** theory_lib, int data_size
    kernel<<<Nblock, blockDim>>>(d_best_corr, d_best_offset, d_best_scale, d_experiment, d_theory_lib, trace_length);


    
    float * best_corr   = new float[Nblock];
    float * best_offset = new float[Nblock];
    float * best_scale  = new float[Nblock];
    hipMemcpy(best_corr,   d_best_corr,   sizeof(float) * Nblock, hipMemcpyDeviceToHost);
    hipMemcpy(best_offset, d_best_offset, sizeof(float) * Nblock, hipMemcpyDeviceToHost);
    hipMemcpy(best_scale,  d_best_scale,  sizeof(float) * Nblock, hipMemcpyDeviceToHost);

    hipFree(d_best_corr);
    hipFree(d_best_offset);
    hipFree(d_best_scale);
    hipFree(d_experiment);
    hipFree(d_theory_lib);

    delete[] experiment;
    delete[] theory_lib;
    return 0;
}